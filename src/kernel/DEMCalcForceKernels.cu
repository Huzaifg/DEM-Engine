#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/Defines.h>
#include <kernel/DEMHelperKernels.cu>
#include <kernel/DEMCollisionKernels.cu>

// If clump templates are jitified, they will be below
_clumpTemplateDefs_;
// Definitions of analytical entites are below
_analyticalEntityDefs_;
// Material properties are below
_materialDefs_;
// If mass properties are jitified, then they are below
_massDefs_;

template <typename T1>
inline __device__ void equipOwnerPosRot(deme::DEMDataDT* granData,
                                        const deme::bodyID_t& myOwner,
                                        T1& relPos,
                                        double3& ownerPos,
                                        double3& bodyPos,
                                        float4& oriQ) {
    voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
        ownerPos.x, ownerPos.y, ownerPos.z, granData->voxelID[myOwner], granData->locX[myOwner],
        granData->locY[myOwner], granData->locZ[myOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
    oriQ.w = granData->oriQw[myOwner];
    oriQ.x = granData->oriQx[myOwner];
    oriQ.y = granData->oriQy[myOwner];
    oriQ.z = granData->oriQz[myOwner];
    applyOriQToVector3(relPos.x, relPos.y, relPos.z, oriQ.w, oriQ.x, oriQ.y, oriQ.z);
    bodyPos.x = ownerPos.x + (double)relPos.x;
    bodyPos.y = ownerPos.y + (double)relPos.y;
    bodyPos.z = ownerPos.z + (double)relPos.z;
}

__global__ void calculateContactForces(deme::DEMSimParams* simParams, deme::DEMDataDT* granData, size_t nContactPairs) {
    deme::contactPairs_t myContactID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < nContactPairs) {
        // Identify contact type first
        deme::contact_t myContactType = granData->contactType[myContactID];
        // The following quantities are always calculated, regardless of force model
        double3 contactPnt;
        float3 B2A, AOwnerMOI, BOwnerMOI;  // Unit vector pointing from body B to body A (contact normal)
        double overlapDepth;
        double3 AOwnerPos, bodyAPos, BOwnerPos, bodyBPos;
        float AOwnerMass, ARadius, BOwnerMass, BRadius;
        float4 AOriQ, BOriQ;
        deme::materialsOffset_t bodyAMatType, bodyBMatType;
        deme::bodyID_t AOwner, BOwner;
        // Then allocate the optional quantities that will be needed in the force model (note: this one can't be in a
        // curly bracket, obviously...)
        _forceModelIngredientDefinition_;
        // Take care of 2 bodies in order, bodyA first, grab location and velocity to local cache
        // We know in this kernel, bodyA will be a sphere; bodyB can be something else
        {
            deme::bodyID_t sphereID = granData->idGeometryA[myContactID];
            deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];
            AOwner = myOwner;

            float3 myRelPos;
            float myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                float3 myMOI;
                _massAcqStrat_;
                _moiAcqStrat_;
                AOwnerMass = myMass;
                AOwnerMOI = myMOI;
            }

            equipOwnerPosRot(granData, myOwner, myRelPos, AOwnerPos, bodyAPos, AOriQ);

            ARadius = myRadius;
            bodyAMatType = granData->sphereMaterialOffset[sphereID];

            // Optional force model ingredients are loaded here...
            _forceModelIngredientAcqForA_;
        }

        // Then bodyB, location and velocity
        if (myContactType == deme::SPHERE_SPHERE_CONTACT) {
            deme::bodyID_t sphereID = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];
            BOwner = myOwner;

            float3 myRelPos;
            float myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                float3 myMOI;
                _massAcqStrat_;
                _moiAcqStrat_;
                BOwnerMass = myMass;
                BOwnerMOI = myMOI;
            }

            equipOwnerPosRot(granData, myOwner, myRelPos, BOwnerPos, bodyBPos, BOriQ);

            BRadius = myRadius;
            bodyBMatType = granData->sphereMaterialOffset[sphereID];

            _forceModelIngredientAcqForB_;

            myContactType = checkSpheresOverlap<double, float>(
                bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, bodyBPos.x, bodyBPos.y, bodyBPos.z, BRadius, contactPnt.x,
                contactPnt.y, contactPnt.z, B2A.x, B2A.y, B2A.z, overlapDepth);
        } else if (myContactType == deme::SPHERE_MESH_CONTACT) {
            deme::bodyID_t triB = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = granData->ownerMesh[triB];
            BOwner = myOwner;

            //// TODO: Is this OK?
            BRadius = DEME_HUGE_FLOAT;
            bodyBMatType = granData->triMaterialOffset[triB];

            double3 triNode1 = to_double3(granData->relPosNode1[triB]);
            double3 triNode2 = to_double3(granData->relPosNode2[triB]);
            double3 triNode3 = to_double3(granData->relPosNode3[triB]);

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                float3 myMOI;
                _massAcqStrat_;
                _moiAcqStrat_;
                BOwnerMass = myMass;
                BOwnerMOI = myMOI;
            }

            // bodyBPos is for a place holder for the outcome triNode1 position
            equipOwnerPosRot(granData, myOwner, triNode1, BOwnerPos, bodyBPos, BOriQ);
            triNode1 = bodyBPos;
            // Do this to node 2 and 3 as well
            applyOriQToVector3(triNode2.x, triNode2.y, triNode2.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);
            triNode2 += BOwnerPos;
            applyOriQToVector3(triNode3.x, triNode3.y, triNode3.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);
            triNode3 += BOwnerPos;
            // Assign the correct bodyBPos
            bodyBPos = triangleCentroid<double3>(triNode1, triNode2, triNode3);

            _forceModelIngredientAcqForB_;

            double3 contact_normal;
            bool in_contact = triangle_sphere_CD<double3, double>(triNode1, triNode2, triNode3, bodyAPos, ARadius,
                                                                  contact_normal, overlapDepth, contactPnt);
            B2A = to_float3(contact_normal);
            overlapDepth = -overlapDepth;  // triangle_sphere_CD gives neg. number for overlapping cases

            // If not in contact, correct myContactType
            if (!in_contact) {
                myContactType = deme::NOT_A_CONTACT;
            }
        } else {
            // If B is analytical entity, its owner, relative location, material info is jitified
            deme::objID_t bodyB = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = objOwner[bodyB];
            bodyBMatType = objMaterial[bodyB];
            BOwner = myOwner;

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                float3 myMOI;
                _massAcqStrat_;
                _moiAcqStrat_;
                BOwnerMass = myMass;
                BOwnerMOI = myMOI;
            }
            //// TODO: Is this OK?
            BRadius = DEME_HUGE_FLOAT;
            float3 myRelPos;
            float3 bodyBRot;
            myRelPos.x = objRelPosX[bodyB];
            myRelPos.y = objRelPosY[bodyB];
            myRelPos.z = objRelPosZ[bodyB];

            equipOwnerPosRot(granData, myOwner, myRelPos, BOwnerPos, bodyBPos, BOriQ);

            // B's orientation (such as plane normal) is rotated with its owner too
            bodyBRot.x = objRotX[bodyB];
            bodyBRot.y = objRotY[bodyB];
            bodyBRot.z = objRotZ[bodyB];
            applyOriQToVector3<float, deme::oriQ_t>(bodyBRot.x, bodyBRot.y, bodyBRot.z, BOriQ.w, BOriQ.x, BOriQ.y,
                                                    BOriQ.z);

            _forceModelIngredientAcqForB_;

            // Note for this test on dT side we don't enlarge entities
            myContactType = checkSphereEntityOverlap<double3, float, double>(
                bodyAPos, ARadius, objType[bodyB], bodyBPos, bodyBRot, objSize1[bodyB], objSize2[bodyB],
                objSize3[bodyB], objNormal[bodyB], 0.0, contactPnt, B2A, overlapDepth);
        }

        float3 force = make_float3(0, 0, 0);
        float3 torque_only_force = make_float3(0, 0, 0);
        _forceModelContactWildcardAcq_;
        if (myContactType != deme::NOT_A_CONTACT) {
            // Local position of the contact point is always a piece of info we require... regardless of force model
            float3 locCPA = to_float3(contactPnt - AOwnerPos);
            float3 locCPB = to_float3(contactPnt - BOwnerPos);
            // Now map this contact point location to bodies' local ref
            applyOriQToVector3<float, deme::oriQ_t>(locCPA.x, locCPA.y, locCPA.z, AOriQ.w, -AOriQ.x, -AOriQ.y,
                                                    -AOriQ.z);
            applyOriQToVector3<float, deme::oriQ_t>(locCPB.x, locCPB.y, locCPB.z, BOriQ.w, -BOriQ.x, -BOriQ.y,
                                                    -BOriQ.z);
            // The following part, the force model, is user-specifiable
            // NOTE!! "force" and "delta_tan" and "delta_time" must be properly set by this piece of code
            { _DEMForceModel_; }

            // Write contact location values back to global memory
            // granData->contactPointGeometryA[myContactID] = locCPA;
            // granData->contactPointGeometryB[myContactID] = locCPB;

            // Take care of A
            {
                atomicAdd(granData->aX + AOwner, force.x / AOwnerMass);
                atomicAdd(granData->aY + AOwner, force.y / AOwnerMass);
                atomicAdd(granData->aZ + AOwner, force.z / AOwnerMass);

                // torque_inForceForm is usually the contribution of rolling resistance and it contributes to torque
                // only, not linear velocity
                float3 myF = (force + torque_only_force);
                // F is in global frame, but it needs to be in local to coordinate with moi and cntPnt
                applyOriQToVector3<float, deme::oriQ_t>(myF.x, myF.y, myF.z, AOriQ.w, -AOriQ.x, -AOriQ.y, -AOriQ.z);
                const float3 angAcc = cross(locCPA, myF) / AOwnerMOI;
                atomicAdd(granData->alphaX + AOwner, angAcc.x);
                atomicAdd(granData->alphaY + AOwner, angAcc.y);
                atomicAdd(granData->alphaZ + AOwner, angAcc.z);
            }

            // Take care of B
            {
                atomicAdd(granData->aX + BOwner, -force.x / BOwnerMass);
                atomicAdd(granData->aY + BOwner, -force.y / BOwnerMass);
                atomicAdd(granData->aZ + BOwner, -force.z / BOwnerMass);

                // torque_inForceForm is usually the contribution of rolling resistance and it contributes to torque
                // only, not linear velocity
                float3 myF = (force + torque_only_force);
                // F is in global frame, but it needs to be in local to coordinate with moi and cntPnt
                applyOriQToVector3<float, deme::oriQ_t>(myF.x, myF.y, myF.z, BOriQ.w, -BOriQ.w, -BOriQ.y, -BOriQ.z);
                const float3 angAcc = cross(locCPB, -myF) / BOwnerMOI;
                atomicAdd(granData->alphaX + BOwner, angAcc.x);
                atomicAdd(granData->alphaY + BOwner, angAcc.y);
                atomicAdd(granData->alphaZ + BOwner, angAcc.z);
            }

        } else {
            // The contact is no longer active, so we need to destroy its contact history recording
            _forceModelContactWildcardDestroy_;
        }
        // granData->contactForces[myContactID] = force;
        // granData->contactTorque_convToForce[myContactID] = torque_only_force;

        // Updated contact wildcards need to be write back to global mem
        _forceModelContactWildcardWrite_;
    }
}
